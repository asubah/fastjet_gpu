#include "hip/hip_runtime.h"
#include <cmath>

#include <hip/hip_runtime.h>

#include "PseudoJet.h"
#include "cluster.h"
#include "cudaCheck.h"

using namespace std;

#pragma region consts
const double pi = 3.141592653589793238462643383279502884197;
const double twopi = 6.283185307179586476925286766559005768394;
const double MaxRap = 1e5;
const double R = 0.6;
const double R2 = R * R;
const double invR2 = 1.0 / R2;
// const double MAX_DOUBLE = 1.79769e+308;
const int grid_max_x = 50;
const int grid_max_y = twopi / R + 1;
const int eta_offsit = 25;
#pragma endregion

#pragma region struct
struct EtaPhi {
  double eta;
  double phi;
  double diB;
  int box_i;
  int box_j;
};

struct Dist {
  double distance;
  int i;
  int j;
};
#pragma endregion

#pragma region device_functions
__host__ __device__ EtaPhi _set_jet(PseudoJet &jet) {
  EtaPhi point;

  point.diB = jet.px * jet.px + jet.py * jet.py;
  jet.isJet = false;

  if (point.diB == 0.0) {
    point.phi = 0.0;
  } else {
    point.phi = atan2(jet.py, jet.px);
  }
  if (point.phi < 0.0) {
    point.phi += twopi;
  }
  if (point.phi >= twopi) {
    point.phi -= twopi;
  } // can happen if phi=-|eps<1e-15|?
  if (jet.E == abs(jet.pz) && point.diB == 0) {
    // Point has infinite rapidity -- convert that into a very large
    // number, but in such a way that different 0-pt momenta will have
    // different rapidities (so as to lift the degeneracy between
    // them) [this can be relevant at parton-level]
    double MaxRapHere = MaxRap + abs(jet.pz);
    if (jet.pz >= 0.0) {
      point.eta = MaxRapHere;
    } else {
      point.eta = -MaxRapHere;
    }
  } else {
    // get the rapidity in a way that's modestly insensitive to roundoff
    // error when things pz,E are large (actually the best we can do without
    // explicit knowledge of mass)
    double effective_m2 = max(0.0, (jet.E + jet.pz) * (jet.E - jet.pz) -
                                       point.diB); // force non tachyonic mass
    double E_plus_pz = jet.E + abs(jet.pz);        // the safer of p+, p-
    // p+/p- = (p+ p-) / (p-)^2 = (kt^2+m^2)/(p-)^2
    point.eta = 0.5 * log((point.diB + effective_m2) / (E_plus_pz * E_plus_pz));
    if (jet.pz > 0) {
      point.eta = -point.eta;
    }
  }

  return point;
}

__device__ double plain_distance(const EtaPhi &p1, const EtaPhi &p2) {
  double dphi = abs(p1.phi - p2.phi);
  if (dphi > pi) {
    dphi = twopi - dphi;
  }
  double drap = p1.eta - p2.eta;
  return (dphi * dphi + drap * drap);
}

__device__ Dist yij_distance(const EtaPhi *points, int i, int j) {
  // k is the one in qusetion
  // d k tid
  if (i > j) {
    int t = i;
    i = j;
    j = t;
  }

  Dist d;
  d.i = i;
  d.j = j;
  // if k == tid return diB
  if (i == j)
    d.distance = points[i].diB;
  else
    d.distance = min(points[i].diB, points[j].diB) *
                 plain_distance(points[i], points[j]) * invR2;

  return d;
}

__device__ Dist minimum_in_cell(const int *grid, const EtaPhi *points, const PseudoJet *jets,
                                Dist min, const int tid, const int i, const int j, const int n) {
  int k = 0;
  int offset = (j * n) + (i * grid_max_y * n);
  int num = grid[offset + k];

  // PseudoJet jet1 = jets[tid];
  // PseudoJet jet2;
  Dist temp;
  while (num > 0) {
    if (tid != num) {
      temp = yij_distance(points, tid, num);

      if (temp.distance < min.distance)
        min = temp;
    }

    k++;
    num = grid[offset + k];
  }

  return min;
}

__device__ void remove_from_grid(int *grid, PseudoJet &jet, const EtaPhi &p, const int n) {
  // Remove from grid
  int k = 0;
  int offset = (p.box_j * n) + (p.box_i * grid_max_y * n);
  int num = grid[offset + k];
  bool shift = false;

  while (num != -1) {
    if (jet.index == num)
      shift = true;
    if (shift) {
      grid[offset + k] = grid[offset + k + 1];
    }
    k++;

    num = grid[offset + k];
  }
}

__device__ void add_to_grid(int *grid, const PseudoJet &jet, const EtaPhi &p, const int n) {
  // Remove from grid
  int k = 0;
  int offset = (p.box_j * n) + (p.box_i * grid_max_y * n);
  int num = grid[offset + k];

  while (true) {
    num = grid[offset + k];
    if (num == -1) {
      grid[offset + k] = jet.index;
      grid[offset + k + 1] = -1;
      break;
    }
    k++;
  }
}
#pragma endregion

#pragma region kernels
__global__ void set_points(PseudoJet *jets, EtaPhi *points, const int n, const float r) {
  int tid = threadIdx.x;

  if (tid >= n)
    return;

  EtaPhi p = _set_jet(jets[tid]);
  p.box_i = p.eta / r + eta_offsit;
  p.box_j = p.phi / r;

  points[tid] = p;
}

__global__ void set_grid(int *grid, const EtaPhi *points, const PseudoJet *jets, const int n) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  // printf("%4d%4d: ", bid, tid);

  int k = 0;
  EtaPhi p;

  int offset = (tid * n) + (bid * grid_max_y * n);

  // if (bid == 0)
  //   printf("%4d%4d%10d\n", bid, tid, offset);
  for (int i = 0; i < n; i++) {
    p = points[i];

    if (p.box_i == bid && p.box_j == tid) {
      grid[offset + k] = jets[i].index;
      // printf("%4d%4d%4d\n", bid, tid, grid[bid * 64 + tid * 64 +
      // k]);
      k++;
    }
  }

  // if (bid)

  grid[offset + k] = -1;
  // printf("-1\n");
}

__global__ void reduce_recombine(int *grid, EtaPhi *points, PseudoJet *jets,
                                 Dist *min_dists, int n, const float r, const int N) {
  extern __shared__ Dist sdata[];

  int tid = threadIdx.x;

  if (tid >= n)
    return;

  min_dists[tid].i = -3;
  min_dists[tid].j = -1;
  Dist min;
  min.i = -4;
  min.j = -4;
  while (n > 0) {

    if (tid >= n)
      return;

    EtaPhi p = points[tid];
    Dist local_min = min_dists[tid];
    if (local_min.i == -3 || local_min.j == min.i || local_min.j == min.j ||
        local_min.i == min.i || local_min.i == min.j || local_min.i >= n ||
        local_min.j >= n) {

      EtaPhi bp;

      min = yij_distance(points, tid, tid);
#define NODY false
#if defined NODY
      // printf("not dynamic!\n");

      min = minimum_in_cell(grid, points, jets, min, tid, p.box_i, p.box_j, N);

      bool right = true;
      bool left = true;
      bool up = true;
      bool down = true;

      bp.eta = ((p.box_i + 1 - eta_offsit) * r);
      bp.phi = p.phi;
      if (min.distance < plain_distance(p, bp)) {
        // printf("saved right!\n");
        right = false;
      }

      bp.eta = ((p.box_i - eta_offsit) * r);
      bp.phi = p.phi;
      if (min.distance < plain_distance(p, bp)) {
        // printf("saved left!\n");
        // printf("%20.8e\n", bp.eta);
        // printf("%20.8e\n", points[min.j].eta);
        left = false;
      }

      bp.eta = p.eta;
      bp.phi = p.box_j + 1 <= grid_max_y ? (p.box_j + 1) * r : 0;
      if (min.distance < plain_distance(p, bp)) {
        // printf("saved up!\n");
        up = false;
      }

      bp.eta = p.eta;
      bp.phi = p.box_j - 1 >= 0 ? p.box_j * r : (grid_max_y - 1) * r;
      if (min.distance < plain_distance(p, bp) && p.box_j - 1 >= 0) {
        // printf("saved down!\n");
        down = false;
      }

      // Right
      if (p.box_i + 1 < grid_max_x + 1 && right) {
        min = minimum_in_cell(grid, points, jets, min, tid, p.box_i + 1,
                              p.box_j, N);
      }

      // Left
      if (p.box_i - 1 >= 0 && left) {
        min = minimum_in_cell(grid, points, jets, min, tid, p.box_i - 1,
                              p.box_j, N);
      }

      // check if above grid_max_y
      int j = p.box_j + 1 <= grid_max_y ? p.box_j + 1 : 0;

      // Up
      if (up) {
        min = minimum_in_cell(grid, points, jets, min, tid, p.box_i, j, N);

        // Up Right
        if (p.box_i + 1 < grid_max_x + 1 && right) {
          min =
              minimum_in_cell(grid, points, jets, min, tid, p.box_i + 1, j, N);
        }

        // Up Left
        if (p.box_i - 1 >= 0 && left) {
          min =
              minimum_in_cell(grid, points, jets, min, tid, p.box_i - 1, j, N);
        }
      }

      // check if bellow 0
      j = p.box_j - 1 >= 0 ? p.box_j - 1 : grid_max_y - 1;

      if (down) {
        // Down
        min = minimum_in_cell(grid, points, jets, min, tid, p.box_i, j, N);

        // Down Right
        if (p.box_i + 1 < grid_max_x + 1 && right) {
          min =
              minimum_in_cell(grid, points, jets, min, tid, p.box_i + 1, j, N);
        }

        // Down Left
        if (p.box_i - 1 >= 0 && left) {
          min =
              minimum_in_cell(grid, points, jets, min, tid, p.box_i - 1, j, N);
        }

        if (p.box_j - 1 < 0) {
          // Down Down
          min =
              minimum_in_cell(grid, points, jets, min, tid, p.box_i, j - 1, N);

          // Down Down Right
          if (p.box_i + 1 < grid_max_x + 1 && right) {
            min = minimum_in_cell(grid, points, jets, min, tid, p.box_i + 1,
                                  j - 1, N);
          }

          // Down Down Left
          if (p.box_i - 1 >= 0 && left) {
            min = minimum_in_cell(grid, points, jets, min, tid, p.box_i - 1,
                                  j - 1, N);
          }
        }
      }
#endif

#if !defined NODY
      minimum_in_cell_kernel<<<1, 12, 12 * sizeof(Dist)>>>(
          grid, points, jets, min_dists, min, tid, p, N);
      hipDeviceSynchronize();
#endif

      int t;
      if (min.i > min.j) {
        t = min.i;
        min.i = min.j;
        min.j = t;
      }

#if defined NODY
      min_dists[tid] = min;
#endif
    }

    sdata[tid] = min_dists[tid];
    __syncthreads();

    for (unsigned int s = 256; s > 0; s >>= 1) {
      if (tid < s && (tid + s) < n) {
        if (sdata[tid + s].distance < sdata[tid].distance) {
          sdata[tid] = sdata[tid + s];
        }
      }
      __syncthreads();
    }

    // Minimum of the row
    // if (tid == 0) {
    // min_dists[k] = sdata[0];
    // }

    min = sdata[0];
    if (tid == 0) {
      // Dist d = yij_distance(points, 57, 61);
      PseudoJet jet_i, jet_j;

      EtaPhi p1, p2;
      if (min.i == min.j) {
        jet_j = jets[min.j];
        p1 = points[min.j];
        remove_from_grid(grid, jet_j, p1, N);
        if (min.j != n - 1)
          remove_from_grid(grid, jets[n - 1], points[n - 1], N);

        jets[min.j] = jets[n - 1];
        points[min.j] = points[n - 1];

        jets[min.j].index = min.j;

        jet_j.isJet = true;
        jet_j.index = n - 1;
        jets[n - 1] = jet_j;
        points[n - 1] = p1;

        if (min.j != n - 1)
          add_to_grid(grid, jets[min.j], points[min.j], N);

      } else {
        jet_i = jets[min.i];
        jet_j = jets[min.j];

        remove_from_grid(grid, jet_i, points[min.i], N);
        remove_from_grid(grid, jet_j, points[min.j], N);
        if (min.j != n - 1) {
          // printf("removing: %4d\n", n - 1);
          remove_from_grid(grid, jets[n - 1], points[n - 1], N);
        }

        jet_i.px += jet_j.px;
        jet_i.py += jet_j.py;
        jet_i.pz += jet_j.pz;
        jet_i.E += jet_j.E;
        p2 = _set_jet(jet_i);

        p2.box_i = p2.eta / r + eta_offsit;
        p2.box_j = p2.phi / r;

        jet_i.index = min.i;

        jets[min.i] = jet_i;
        points[min.i] = p2;

        jets[min.j] = jets[n - 1];
        points[min.j] = points[n - 1];
        jets[min.j].index = min.j;

        add_to_grid(grid, jet_i, p2, N);
        if (min.j != n - 1)
          add_to_grid(grid, jets[min.j], points[min.j], N);
      }
    }
    n--;
    __syncthreads();
  }
}
#pragma endregion


void cluster(PseudoJet* particles, int size) {
#pragma region vectors
  EtaPhi* d_points_ptr;
  cudaCheck(hipMalloc(&d_points_ptr, sizeof(EtaPhi) * size));

  int *d_grid_ptr;
  cudaCheck(hipMalloc(&d_grid_ptr, sizeof(int) * size * grid_max_x * grid_max_y));

  Dist *d_min_dists_ptr;
  cudaCheck(hipMalloc(&d_min_dists_ptr, sizeof(Dist) * size));
#pragma endregion

#pragma region kernel_launches
  // set jets into points
  set_points<<<1, 512>>>(particles, d_points_ptr, size, R);

  // create grid
  set_grid<<<grid_max_x + 1, grid_max_y>>>(d_grid_ptr, d_points_ptr,
      particles, size);

  // compute dist_min
  // for (int i = n; i > 0; i--) {
  // compute_nn<<<1, n>>>(d_grid_ptr, d_points_ptr, particles,
  //                      d_min_dists_ptr, i, N);

  reduce_recombine<<<1, 354, sizeof(Dist) * size>>>(
      d_grid_ptr, d_points_ptr, particles, d_min_dists_ptr, size, R, size);
#pragma endregion
}
