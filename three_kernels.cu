
#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>
#include <cmath>
#include <assert.h>
#include <stdio.h>

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

using namespace std;

struct PseudoJet
{
    double px;
    double py;
    double pz;
    double E;
    double diB;
    double phi;
    double rap;
};

const double pi = 3.141592653589793238462643383279502884197;
const double twopi = 6.283185307179586476925286766559005768394;
const double MaxRap = 1e5;
const double R = 0.6;
const double R2 = R * R;
const double invR2 = 1.0 / R2;
const double ptmin = 5.0;
const double dcut = ptmin * ptmin;
int const NUM_PARTICLES = 354;

__device__ void _set_jet(PseudoJet &jet)
{
    // printf("%15.8e %15.8e", jet.px, jet.py);
    jet.diB = jet.px * jet.px + jet.py * jet.py;

    if (jet.diB == 0.0)
    {
        jet.phi = 0.0;
    }
    else
    {
        jet.phi = atan2(jet.py, jet.px);
    }
    if (jet.phi < 0.0)
    {
        jet.phi += twopi;
    }
    if (jet.phi >= twopi)
    {
        jet.phi -= twopi;
    } // can happen if phi=-|eps<1e-15|?
    if (jet.E == abs(jet.pz) && jet.diB == 0)
    {
        // Point has infinite rapidity -- convert that into a very large
        // number, but in such a way that different 0-pt momenta will have
        // different rapidities (so as to lift the degeneracy between
        // them) [this can be relevant at parton-level]
        double MaxRapHere = MaxRap + abs(jet.pz);
        if (jet.pz >= 0.0)
        {
            jet.rap = MaxRapHere;
        }
        else
        {
            jet.rap = -MaxRapHere;
        }
    }
    else
    {
        // get the rapidity in a way that's modestly insensitive to roundoff
        // error when things pz,E are large (actually the best we can do without
        // explicit knowledge of mass)
        double effective_m2 = max(0.0, (jet.E + jet.pz) * (jet.E - jet.pz) - jet.diB); // force non tachyonic mass
        double E_plus_pz = jet.E + abs(jet.pz);                                        // the safer of p+, p-
        // p+/p- = (p+ p-) / (p-)^2 = (kt^2+m^2)/(p-)^2
        jet.rap = 0.5 * log((jet.diB + effective_m2) / (E_plus_pz * E_plus_pz));
        if (jet.pz > 0)
        {
            jet.rap = -jet.rap;
        }
    }
}

__device__ double plain_distance(PseudoJet &jet1, PseudoJet &jet2)
{
    double dphi = abs(jet1.phi - jet2.phi);
    if (dphi > pi)
    {
        dphi = twopi - dphi;
    }
    double drap = jet1.rap - jet2.rap;
    return (dphi * dphi + drap * drap);
}

__device__ double yij_distance(PseudoJet &jet1, PseudoJet &jet2)
{
    return min(jet1.diB, jet2.diB) *
           plain_distance(jet1, jet2) *
           invR2;
}

__device__ void tid_to_ij(int &i, int &j, int tid, int n, int N)
{
    int ii = N - 1 - tid;
    int k = floor((sqrt(8.0 * ii + 1) - 1) / 2);
    i = n - 1 - k;
    j = tid - N + ((k + 1) * (k + 2) / 2) + i;
}

double plain_distance_h(PseudoJet &jet1, PseudoJet &jet2)
{
    double dphi = abs(jet1.phi - jet2.phi);
    if (dphi > pi)
    {
        dphi = twopi - dphi;
    }
    double drap = jet1.rap - jet2.rap;
    return (dphi * dphi + drap * drap);
}

double yij_distance_h(PseudoJet &jet1, PseudoJet &jet2)
{
    return min(jet1.diB, jet2.diB) *
           plain_distance_h(jet1, jet2) *
           invR2;
}

__global__ void set_jets(PseudoJet *jets)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    _set_jet(jets[tid]);
    // __syncthreads();
    // if(tid == 0)
    //     for(int i = 0; i < NUM_PARTICLES; i++)
    //         printf("%10.8f%10.8f%10.8f%10.8f%10.8f%10.8f%10.8f\n",
    //             jets[i].px,
    //             jets[i].py,
    //             jets[i].pz,
    //             jets[i].E,
    //             jets[i].diB,
    //             jets[i].phi,
    //             jets[i].rap
    //         );
}

__global__ void set_distances(PseudoJet *jets, double *distances,
                              int *indices, int const num_particles)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int N = num_particles * (num_particles + 1) / 2;

    if (tid >= N)
        return;

    indices[tid] = tid;

    int i, j;
    tid_to_ij(i, j, tid, num_particles, N);

    if (i == j)
    {
        distances[tid] = jets[i].diB;
    }
    else
    {
        distances[tid] = yij_distance(jets[i], jets[j]);
        // if (distances[tid] <= 0.0000003)
        // printf("i = %d j = %d tid = %d d = %15.8e\n", i, j, tid, distances[tid]);
    }
    // __syncthreads();

    // if (tid == 0)
    //     for (int tid = 0; tid < gridDim.x * blockDim.x; tid++)
    //     {
    //         int i = tid / NUM_PARTICLES;
    //         int j = (tid % NUM_PARTICLES) - 1;

    //         if (i == j)
    //         {
    //             // cout << endl
    //             //      << h_distances[tid];
    //             if (distances[tid] == jets[i].diB)
    //                 printf("\n0");
    //             else
    //                 printf("\n1");
    //         }
    //         else
    //         {
    //             // cout << endl
    //             //      << h_distances[tid];
    //             if (distances[tid] == yij_distance(jets[i], jets[j]))
    //                 printf(" 0");
    //             else
    //                 printf(" 1");
    //         }
    //     }
    // for (int i = 0; i < gridDim.x * blockDim.x; i++)
    // printf("%d %10.5f\n", tid, distances[tid]);
}

__global__ void recalculate_distances(PseudoJet *jets, double *distances,
                                      int const num_particles,
                                      int const row, int const column)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int i = num_particles * tid + column;

    if (tid > row)
        return;

     printf("tid = %d, column = %d\n", tid, column);

    if (tid == column)
        distances[i] = jets[tid].diB;
    else
        distances[i] = yij_distance(jets[tid], jets[column]);
}

__global__ void reduction_min(PseudoJet *jets, double *distances, double *out, int *indices,
                              int const num_particles, int const memory_size,
                              int const array_size, bool const first)
{
    // int N = num_particles * (num_particles + 1) / 2;
    extern __shared__ double sdata[];
    double *s_distances = sdata;
    int *s_indices = (int *)&s_distances[memory_size];

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i = threadIdx.x;

    if (tid >= num_particles)
        return;

    s_distances[i] = distances[tid];

    if (first)
        s_indices[i] = tid;
    else
        s_indices[i] = indices[tid];
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (i < s && (tid + s) < num_particles)
        {
            if (s_distances[i] > s_distances[i + s])
            {
                s_distances[i] = s_distances[i + s];

                s_indices[i] = s_indices[i + s];
            }
        }
        __syncthreads();
    }

    if (i == 0)
    {
        out[blockIdx.x] = s_distances[0];
        int min_tid = s_indices[0];
        indices[blockIdx.x] = min_tid;

        // printf("d = %20.17f i = %d\n", s_distances[0], s_indices[0]);
        if (!first)
        {
            int N = array_size * (array_size + 1) / 2;
            int i, j;
            tid_to_ij(i, j, min_tid, array_size, N);
            printf("i = %d, j = %d\n", i, j);

            if (i == j)
            {
                jets[j] = jets[array_size - 1];
            }
            else
            {
                jets[i].px += jets[j].px;
                jets[i].py += jets[j].py;
                jets[i].pz += jets[j].pz;
                jets[i].E += jets[j].E;
                _set_jet(jets[i]);

                jets[j] = jets[array_size - 1];
            }

            if (j + 1 < 1024)
                recalculate_distances<<<1, (j + 1)>>>(
                    jets, distances, array_size, i, j);
            else
            {
                int num_blocks = ((j + 1) / 1024) + 1;
                recalculate_distances<<<num_blocks, 1024>>>(
                    jets, distances, array_size, i, j);
            }
        }
    }
}


int main()
{
    int d_id;
    hipDeviceProp_t d_prop;

    hipChooseDevice(&d_id, &d_prop);
    cout << "device id is " << d_id << endl;
    hipSetDevice(d_id);

    PseudoJet *h_jets = 0;
    h_jets = (PseudoJet *)malloc(NUM_PARTICLES * sizeof(PseudoJet));

    double *h_mini = 0;
    h_mini = (double *)malloc(sizeof(double));
    *h_mini = numeric_limits<double>::max();

    int i;
    for (i = 0; i < NUM_PARTICLES; i++)
    {
        cin >> h_jets[i].px >> h_jets[i].py >> h_jets[i].pz >> h_jets[i].E;
    }

    PseudoJet *d_jets = 0;
    hipMalloc((void **)&d_jets, NUM_PARTICLES * sizeof(PseudoJet));
    hipMemcpy(d_jets, h_jets, NUM_PARTICLES * sizeof(PseudoJet), hipMemcpyHostToDevice);

    double *d_distances = 0;
    hipMalloc((void **)&d_distances,
               (NUM_PARTICLES * (NUM_PARTICLES + 1) / 2) * sizeof(double));

    int *d_indices = 0;
    hipMalloc((void **)&d_indices,
               (NUM_PARTICLES * (NUM_PARTICLES + 1) / 2) * sizeof(int));

    double *d_mini = 0;
    hipMalloc((void **)&d_mini, sizeof(double));
    hipMemcpy(d_mini, h_mini, sizeof(double), hipMemcpyHostToDevice);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy calls1");

    int num_threads = 354;
    int num_blocks = (NUM_PARTICLES + num_threads) / (num_threads + 1);
    //std::cout << "blocks = " << num_blocks;
    cout << num_threads << " " << num_blocks << endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    set_jets<<<num_blocks, num_threads>>>(d_jets);

    num_threads = (NUM_PARTICLES * (NUM_PARTICLES + 1) / 2);
    num_blocks = (num_threads / 1024) + 1;
    cout << num_threads << " " << num_blocks << endl;
    set_distances<<<num_blocks, 1024>>>(d_jets, d_distances, d_indices,
                                        NUM_PARTICLES);

    double *d_out = 0;
    hipMalloc((void **)&d_out, num_blocks * sizeof(double));
    for (int n = NUM_PARTICLES; n > 0; n--)
    {
        num_threads = (n * (n + 1) / 2);
        num_blocks = (num_threads / 1024) + 1;
        // cout << num_threads << " " << num_blocks << endl;
        reduction_min<<<num_blocks, 1024,
                        1024 * sizeof(double) + 1024 * sizeof(int)>>>(
            d_jets,
            d_distances,
            d_out,
            d_indices,
            num_threads,
            1024,
            n,
            true);

        reduction_min<<<1, 64,
                        num_blocks * sizeof(double) + num_blocks * sizeof(int)>>>(
            d_jets,
            d_out,
            d_out,
            d_indices,
            num_blocks,
            num_blocks,
            n,
            false);
    }
    hipEventRecord(stop);

    // Check for any CUDA errors
    checkCUDAError("kernal launch");
    hipMemcpy(h_jets, d_jets,
               NUM_PARTICLES * sizeof(PseudoJet),
               hipMemcpyDeviceToHost);

    double *h_out = 0;
    h_out = (double *)malloc(num_blocks * sizeof(double));
    hipMemcpy(h_out, d_out, num_blocks * sizeof(double),
               hipMemcpyDeviceToHost);
    int *h_indices = 0;
    h_indices = (int *)malloc(num_threads * sizeof(int));
    hipMemcpy(h_indices, d_indices, num_threads * sizeof(int),
               hipMemcpyDeviceToHost);
    // for(int i = 0; i < num_blocks; i++)
    // cout << h_out[0] << endl;
    // cout << h_indices[0] << endl;

    for (int i = 0; i < NUM_PARTICLES; i++)
        if (h_jets[i].diB > dcut)
            printf("%15.8f %15.8f %15.8f\n",
                   h_jets[i].rap, h_jets[i].phi, sqrt(h_jets[i].diB));

    // int ii = num_threads - 1 - h_indices[0];
    // int k = floor((sqrt(8.0 * ii + 1) - 1) / 2);
    // int r = NUM_PARTICLES - 1 - k;
    // int c = h_indices[0] - num_threads + ((k + 1) * (k + 2) / 2) + r;
    // cout << r << " " << c << endl;
    // cout << yij_distance_h(h_jets[r], h_jets[c]) << endl;

    // for (int tid = 0; tid < num_threads; tid++)
    // {
    //     int i = tid / NUM_PARTICLES;
    //     int j = (tid % NUM_PARTICLES) - 1;

    //     cout << h_distances[tid] << endl;
    //     // if (i == j)
    //     // {
    //     //     cout << endl
    //     //          << h_distances[tid];
    //     //     // if (h_distances[tid] == h_jets[i].diB)
    //     //     //     cout << "\n0";
    //     //     // else
    //     //     //     cout << "\n1";
    //     // }
    //     // else
    //     // {
    //     //     cout << endl
    //     //          << h_distances[tid];
    //     //     // if (h_distances[tid] == yij_distance_h(h_jets[i], h_jets[j]))
    //     //     //     cout << " 0";
    //     //     // else
    //     //     //     cout << " 1";
    //     // }
    // }

    // Check for any CUDA errors
    checkCUDAError("cudaMemcpy2 calls");

    hipMemcpy(h_mini, d_mini, sizeof(double), hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("cudaMemcpy3 calls");

    //    cout << "d_mini = " << *h_mini << std::endl;
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time = %10.8f\n", milliseconds);

    // free device memory
    hipFree(d_jets);
    hipFree(d_mini);
    hipFree(d_distances);

    // free host memory
    free(h_jets);
    free(h_mini);
    free(h_out);

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
