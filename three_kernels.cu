#include "hip/hip_runtime.h"
#include <assert.h>
#include <cmath>
#include <iostream>
#include <limits>
#include <stdio.h>

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);
bool double_equals(double a, double b, double epsilon = 1e-6);

using namespace std;

struct PseudoJet {
  double px;
  double py;
  double pz;
  double E;
  double diB;
  double phi;
  double rap;
  bool isJet;
};

const double pi = 3.141592653589793238462643383279502884197;
const double twopi = 6.283185307179586476925286766559005768394;
const double MaxRap = 1e5;
const double R = 0.6;
const double R2 = R * R;
const double invR2 = 1.0 / R2;
const double ptmin = 5.0;
const double dcut = ptmin * ptmin;

__device__ void _set_jet(PseudoJet &jet) {
  jet.diB = jet.px * jet.px + jet.py * jet.py;
  jet.isJet = false;

  if (jet.diB == 0.0) {
    jet.phi = 0.0;
  } else {
    jet.phi = atan2(jet.py, jet.px);
  }
  if (jet.phi < 0.0) {
    jet.phi += twopi;
  }
  if (jet.phi >= twopi) {
    jet.phi -= twopi;
  } // can happen if phi=-|eps<1e-15|?
  if (jet.E == abs(jet.pz) && jet.diB == 0) {
    // Point has infinite rapidity -- convert that into a very large
    // number, but in such a way that different 0-pt momenta will have
    // different rapidities (so as to lift the degeneracy between
    // them) [this can be relevant at parton-level]
    double MaxRapHere = MaxRap + abs(jet.pz);
    if (jet.pz >= 0.0) {
      jet.rap = MaxRapHere;
    } else {
      jet.rap = -MaxRapHere;
    }
  } else {
    // get the rapidity in a way that's modestly insensitive to roundoff
    // error when things pz,E are large (actually the best we can do without
    // explicit knowledge of mass)
    double effective_m2 = max(0.0, (jet.E + jet.pz) * (jet.E - jet.pz) -
                                       jet.diB); // force non tachyonic mass
    double E_plus_pz = jet.E + abs(jet.pz);      // the safer of p+, p-
    // p+/p- = (p+ p-) / (p-)^2 = (kt^2+m^2)/(p-)^2
    jet.rap = 0.5 * log((jet.diB + effective_m2) / (E_plus_pz * E_plus_pz));
    if (jet.pz > 0) {
      jet.rap = -jet.rap;
    }
  }
}

void _set_jet_h(PseudoJet &jet) {
  jet.diB = jet.px * jet.px + jet.py * jet.py;
  jet.isJet = false;

  if (jet.diB == 0.0) {
    jet.phi = 0.0;
  } else {
    jet.phi = atan2(jet.py, jet.px);
  }
  if (jet.phi < 0.0) {
    jet.phi += twopi;
  }
  if (jet.phi >= twopi) {
    jet.phi -= twopi;
  } // can happen if phi=-|eps<1e-15|?
  if (jet.E == abs(jet.pz) && jet.diB == 0) {
    // Point has infinite rapidity -- convert that into a very large
    // number, but in such a way that different 0-pt momenta will have
    // different rapidities (so as to lift the degeneracy between
    // them) [this can be relevant at parton-level]
    double MaxRapHere = MaxRap + abs(jet.pz);
    if (jet.pz >= 0.0) {
      jet.rap = MaxRapHere;
    } else {
      jet.rap = -MaxRapHere;
    }
  } else {
    // get the rapidity in a way that's modestly insensitive to roundoff
    // error when things pz,E are large (actually the best we can do without
    // explicit knowledge of mass)
    double effective_m2 = max(0.0, (jet.E + jet.pz) * (jet.E - jet.pz) -
                                       jet.diB); // force non tachyonic mass
    double E_plus_pz = jet.E + abs(jet.pz);      // the safer of p+, p-
    // p+/p- = (p+ p-) / (p-)^2 = (kt^2+m^2)/(p-)^2
    jet.rap = 0.5 * log((jet.diB + effective_m2) / (E_plus_pz * E_plus_pz));
    if (jet.pz > 0) {
      jet.rap = -jet.rap;
    }
  }
}

__device__ double plain_distance(PseudoJet &jet1, PseudoJet &jet2) {
  double dphi = abs(jet1.phi - jet2.phi);
  if (dphi > pi) {
    dphi = twopi - dphi;
  }
  double drap = jet1.rap - jet2.rap;
  return (dphi * dphi + drap * drap);
}

__device__ double yij_distance(PseudoJet &jet1, PseudoJet &jet2) {
  return min(jet1.diB, jet2.diB) * plain_distance(jet1, jet2) * invR2;
}

__device__ void tid_to_ij(int &i, int &j, int tid) {
  tid += 1;
  j = std::ceil(std::sqrt(2 * tid + 0.25) - 0.5);
  i = trunc(tid - (j - 1) * j / 2.0);
  j -= 1;
  i -= 1;
}

__global__ void set_jets(PseudoJet *jets) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  _set_jet(jets[tid]);
}

__global__ void set_distances(PseudoJet *jets, double *distances, int *indices,
                              int *indices_ii, int *indices_jj,
                              int const num_particles) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int N = num_particles * (num_particles + 1) / 2;

  if (tid >= N)
    return;

  indices[tid] = tid;

  int i, j;
  tid_to_ij(i, j, tid);
  indices_ii[tid] = i;
  indices_jj[tid] = j;

  if (i == j) {
    distances[tid] = jets[i].diB;
  } else {
    distances[tid] = yij_distance(jets[i], jets[j]);
  }
}

__global__ void recalculate_distances(PseudoJet *jets, double *distances,
                                      int *indices, int *indices_ii,
                                      int *indices_jj, int const n) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int i, j;
  int index = indices[0];
  // tid_to_ij(i, j, index);
  i = indices_ii[index];
  j = indices_jj[index];

  if (j >= n) {
    // tid_to_ij(i, j, index - n);
    i = indices_ii[index - n];
    j = indices_jj[index - n];
  }
  int tid_j = tid + ((j) * (j + 1) / 2);

  if (tid >= n)
    return;

  if (tid == j) {
    distances[tid_j] = jets[tid].diB;
  } else if (tid < j) {
    distances[tid_j] = yij_distance(jets[tid], jets[j]);
  } else {
    tid_j = j + ((tid) * (tid + 1) / 2);
    distances[tid_j] = yij_distance(jets[tid], jets[j]);
  }

  int tid_i = tid_j;
  if (i != j) {
    tid_i = tid + ((i) * (i + 1) / 2);
    if (tid == i)
      distances[tid_i] = jets[tid].diB;
    else if (tid < i) {
      distances[tid_i] = yij_distance(jets[tid], jets[i]);
    } else {
      tid_i = i + ((tid) * (tid + 1) / 2);
      distances[tid_i] = yij_distance(jets[tid], jets[i]);
    }
  }
}

__global__ void reduction_min_first(PseudoJet *jets, double *distances,
                                    double *distances_out, int *indices,
                                    int *indices_ii, int *indices_jj,
                                    int const distances_array_size,
                                    int const num_particles) {
  extern __shared__ double sdata[];
  double *s_distances = sdata;
  int *s_indices = (int *)&s_distances[blockDim.x];

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int i = threadIdx.x;

  if (tid >= distances_array_size)
    return;

  s_distances[i] = distances[tid];
  s_indices[i] = tid;
  __syncthreads();

  int jj;
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (i < s && (tid + s) < distances_array_size) {
      // tid_to_ij(ii, jj, s_indices[i + s]);
      // ii = indices_ii[s_indices[i + s]];
      jj = indices_jj[s_indices[i + s]];
      if (s_distances[i] > s_distances[i + s] && jj < num_particles) {
        s_distances[i] = s_distances[i + s];

        s_indices[i] = s_indices[i + s];
      }
    }
    __syncthreads();
  }

  if (i == 0) {
    distances_out[blockIdx.x] = s_distances[0];
    int min_tid = s_indices[0];
    indices[blockIdx.x] = min_tid;
  }
}

__global__ void reduction_min_second(PseudoJet *jets, double *distances,
                                     double *distances_out, int *indices,
                                     int *indices_ii, int *indices_jj,
                                     int const distances_array_size,
                                     int const num_particles) {
  extern __shared__ double sdata[];
  double *s_distances = sdata;
  int *s_indices = (int *)&s_distances[blockDim.x];

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int i = threadIdx.x;

  if (tid >= distances_array_size)
    return;

  s_distances[i] = distances[tid];
  s_indices[i] = indices[tid];
  __syncthreads();

  int jj;
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (i < s && (tid + s) < distances_array_size) {
      // tid_to_ij(ii, jj, s_indices[i + s]);
      // ii = indices_ii[s_indices[i + s]];
      jj = indices_jj[s_indices[i + s]];
      if (s_distances[i] > s_distances[i + s] && jj < num_particles) {
        s_distances[i] = s_distances[i + s];

        s_indices[i] = s_indices[i + s];
      }
    }
    __syncthreads();
  }

  if (i == 0) {
    distances_out[blockIdx.x] = s_distances[0];
    int min_tid = s_indices[0];
    indices[blockIdx.x] = min_tid;

    int i, j;
    // tid_to_ij(i, j, min_tid);
    i = indices_ii[min_tid];
    j = indices_jj[min_tid];

    if (i == j) {
      PseudoJet temp;
      temp = jets[j];
      jets[j] = jets[num_particles - 1];
      temp.isJet = true;
      jets[num_particles - 1] = temp;
    } else {
      jets[i].px += jets[j].px;
      jets[i].py += jets[j].py;
      jets[i].pz += jets[j].pz;
      jets[i].E += jets[j].E;
      _set_jet(jets[i]);

      jets[j] = jets[num_particles - 1];
    }
  }
}

int main() {
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  printf("Device Name: %s\n", prop.name);

  int NUM_PARTICLES = 0;
  int NUM_EVENTS = 1;

  for (int event = 0; event < NUM_EVENTS; event++) {
    PseudoJet *h_jets = NULL;
    PseudoJet *h_more_jets = NULL;
    PseudoJet temp;

    NUM_PARTICLES = 0;
    while (true) {
      // h_jets = (PseudoJet *)malloc(NUM_PARTICLES * sizeof(PseudoJet));
      cin >> temp.px >> temp.py >> temp.pz >> temp.E;

      if (cin.fail())
        break;

      NUM_PARTICLES++;

      h_more_jets =
          (PseudoJet *)realloc(h_jets, NUM_PARTICLES * sizeof(PseudoJet));

      if (h_more_jets != NULL) {
        h_jets = h_more_jets;
        h_jets[NUM_PARTICLES - 1] = temp;
      } else {
        free(h_jets);
        puts("Error (re)allocating memory");
        exit(1);
      }
    }

    // if (NUM_PARTICLES != 241)
    //   continue;

    // for (int i = 0; i < NUM_PARTICLES; i++)
    //   cout << h_jets[i].E << endl;

    cin.clear();
    cin.ignore(numeric_limits<streamsize>::max(), '\n');
    // _set_jet_h(h_jets[i]);

    int i;

    for (i = 0; i < NUM_PARTICLES; i++) {
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    PseudoJet *d_jets = 0;
    hipMalloc((void **)&d_jets, NUM_PARTICLES * sizeof(PseudoJet));
    hipMemcpy(d_jets, h_jets, NUM_PARTICLES * sizeof(PseudoJet),
               hipMemcpyHostToDevice);

    double *d_distances = 0;
    hipMalloc((void **)&d_distances,
               (NUM_PARTICLES * (NUM_PARTICLES + 1) / 2) * sizeof(double));

    int *d_indices = 0;
    hipMalloc((void **)&d_indices,
               (NUM_PARTICLES * (NUM_PARTICLES + 1) / 2) * sizeof(int));
    int *d_indices_ii = 0;
    hipMalloc((void **)&d_indices_ii,
               (NUM_PARTICLES * (NUM_PARTICLES + 1) / 2) * sizeof(int));
    int *d_indices_jj = 0;
    hipMalloc((void **)&d_indices_jj,
               (NUM_PARTICLES * (NUM_PARTICLES + 1) / 2) * sizeof(int));

    int num_threads = 354;
    int num_blocks = (NUM_PARTICLES + num_threads) / (num_threads + 1);

    double *d_out = 0;
    hipMalloc((void **)&d_out, num_blocks * sizeof(double));

    vector<double> acc;
    float milliseconds;
    for (int s = 0; s < 1000; s++) {
      hipEventRecord(start);

      set_jets<<<num_blocks, num_threads>>>(d_jets);

      num_threads = (NUM_PARTICLES * (NUM_PARTICLES + 1) / 2);
      num_blocks = (num_threads / 1024) + 1;
      set_distances<<<num_blocks, 1024>>>(d_jets, d_distances, d_indices,
                                          d_indices_ii, d_indices_jj,
                                          NUM_PARTICLES);

      for (int n = NUM_PARTICLES; n > 0; n--) {
        num_threads = (n * (n + 1) / 2);
        num_blocks = (num_threads / 1024) + 1;

        reduction_min_first<<<num_blocks, 1024,
                              1024 * sizeof(double) + 1024 * sizeof(int)>>>(
            d_jets, d_distances, d_out, d_indices, d_indices_ii, d_indices_jj,
            num_threads, n);

        reduction_min_second<<<1, num_blocks, num_blocks * sizeof(double) +
                                                  num_blocks * sizeof(int)>>>(
            d_jets, d_out, d_out, d_indices, d_indices_ii, d_indices_jj,
            num_blocks, n);

        recalculate_distances<<<(NUM_PARTICLES / 1024) + 1, 1024>>>(
            d_jets, d_distances, d_indices, d_indices_ii, d_indices_jj, n - 1);
      }

      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds, start, stop);
      printf("run %d\t%.3fms\n", s, milliseconds);
      acc.push_back(milliseconds);
    }
    hipMemcpy(h_jets, d_jets, NUM_PARTICLES * sizeof(PseudoJet),
               hipMemcpyDeviceToHost);

    // // Check for any CUDA errors
    // checkCUDAError("kernal launch");

    // double *h_out = 0;
    // h_out = (double *)malloc(num_blocks * sizeof(double));
    // hipMemcpy(h_out, d_out, num_blocks * sizeof(double),
    //            hipMemcpyDeviceToHost);

    // // Check for any CUDA errors
    // checkCUDAError("hipMemcpy calls");
    //
    // hipEventSynchronize(stop);

    double sum = std::accumulate(acc.begin(), acc.end(), 0.0);
    double mean = sum / acc.size();

    double sq_sum =
        std::inner_product(acc.begin(), acc.end(), acc.begin(), 0.0);
    double stdev = std::sqrt(sq_sum / acc.size() - mean * mean);
    printf("n =  %d\n", NUM_PARTICLES);
    printf("mean %.3fms\n", mean);
    printf("std %.3fms\n", stdev);

    // for (int i = 0; i < NUM_PARTICLES; i++)
    //   if (h_jets[i].diB >= dcut && h_jets[i].isJet)
    //     printf("%15.8f %15.8f %15.8f\n", h_jets[i].rap, h_jets[i].phi,
    //            sqrt(h_jets[i].diB));

    // free device memory
    hipFree(d_jets);
    hipFree(d_distances);
    hipFree(d_indices);
    hipFree(d_indices_ii);
    hipFree(d_indices_jj);
    hipFree(d_out);

    // free host memory
    free(h_jets);
    // free(h_more_jets);
    // free(h_out);
  }

  return 0;
}

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(-1);
  }
}

bool double_equals(double a, double b, double epsilon) {
  return std::abs(a - b) < epsilon;
}
