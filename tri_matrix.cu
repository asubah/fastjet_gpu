#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <iostream>
#include <limits>
#include <numeric>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <cub/block/block_reduce.cuh>

#include "PseudoJet.h"
#include "cluster.h"
#include "cudaCheck.h"

using namespace std;
using namespace hipcub;

unsigned int upper_power_of_two(int v) {
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;

  return v;
}

struct PseudoJetExt {
  double px;
  double py;
  double pz;
  double E;
  double diB;
  double inv_diB;
  double phi;
  double rap;
  bool isJet;

  __host__ __device__ double get_diB(Scheme scheme) const {
    switch (scheme) {
      case Scheme::Kt:
        return diB;

      case Scheme::CambridgeAachen:
        return 1.;

      case Scheme::AntiKt:
        return inv_diB;
    }
    // never reached
    return diB;
  }
};

const double pi = 3.141592653589793238462643383279502884197;
const double twopi = 6.283185307179586476925286766559005768394;
const double MaxRap = 1e5;
const double MAX_DOUBLE = 1.79769e+308;

__device__ void _set_jet(PseudoJetExt &jet) {
  jet.diB = jet.px * jet.px + jet.py * jet.py;
  jet.inv_diB = jet.diB > 1e-300 ? 1.0 / jet.diB : 1e300;
  jet.isJet = false;

  if (jet.diB == 0.0) {
    jet.phi = 0.0;
  } else {
    jet.phi = atan2(jet.py, jet.px);
  }
  if (jet.phi < 0.0) {
    jet.phi += twopi;
  }
  if (jet.phi >= twopi) {
    jet.phi -= twopi;
  }  // can happen if phi=-|eps<1e-15|?
  if (jet.E == abs(jet.pz) && jet.diB == 0) {
    // Point has infinite rapidity -- convert that into a very large
    // number, but in such a way that different 0-pt momenta will have
    // different rapidities (so as to lift the degeneracy between
    // them) [this can be relevant at parton-level]
    double MaxRapHere = MaxRap + abs(jet.pz);
    if (jet.pz >= 0.0) {
      jet.rap = MaxRapHere;
    } else {
      jet.rap = -MaxRapHere;
    }
  } else {
    // get the rapidity in a way that's modestly insensitive to roundoff
    // error when things pz,E are large (actually the best we can do without
    // explicit knowledge of mass)
    double effective_m2 = max(0.0, (jet.E + jet.pz) * (jet.E - jet.pz) - jet.diB);  // force non tachyonic mass
    double E_plus_pz = jet.E + abs(jet.pz);                                         // the safer of p+, p-
    // p+/p- = (p+ p-) / (p-)^2 = (kt^2+m^2)/(p-)^2
    jet.rap = 0.5 * log((jet.diB + effective_m2) / (E_plus_pz * E_plus_pz));
    if (jet.pz > 0) {
      jet.rap = -jet.rap;
    }
  }
}

__device__ double plain_distance(PseudoJetExt &jet1, PseudoJetExt &jet2) {
  double dphi = abs(jet1.phi - jet2.phi);
  if (dphi > pi) {
    dphi = twopi - dphi;
  }
  double drap = jet1.rap - jet2.rap;
  return (dphi * dphi + drap * drap);
}

__device__ double yij_distance(PseudoJetExt &jet1, PseudoJetExt &jet2, Scheme scheme, double one_over_r2) {
  return min(jet1.get_diB(scheme), jet2.get_diB(scheme)) * plain_distance(jet1, jet2) * one_over_r2;
}

__device__ void tid_to_ij(int &i, int &j, int tid) {
  tid += 1;
  j = std::ceil(std::sqrt(2 * tid + 0.25) - 0.5);
  i = trunc(tid - (j - 1) * j / 2.0);
  j -= 1;
  i -= 1;
}

__global__ void set_jets(PseudoJetExt *jets) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  _set_jet(jets[tid]);
}

struct Dist {
  double d;
  int i;
  int j;
};

struct dist_compare {
  __host__ __device__ Dist operator()(Dist &first, Dist &second) { return first.d < second.d ? first : second; }
};

__global__ void set_distances(
    PseudoJetExt *jets, Dist *distances, Dist *g_min, int num_particles, Scheme scheme, double one_over_r2) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= num_particles)
    return;

  Dist dst;
  tid_to_ij(dst.i, dst.j, tid);

  if (dst.i == dst.j) {
    dst.d = jets[dst.i].get_diB(scheme);
  } else {
    dst.d = yij_distance(jets[dst.i], jets[dst.j], scheme, one_over_r2);
  }

  distances[tid] = dst;

  if (tid == 0) {
    g_min->i = -1;
  }
}

__global__ void reduction_min(PseudoJetExt *jets,
                              Dist *distances,
                              Dist *distances_out,
                              Dist *g_min,
                              int const distances_array_size,
                              int const num_particles,
                              Scheme scheme,
                              double one_over_r2) {
  // Specialize BlockReduce type for our thread block
  typedef BlockReduce<Dist, 1024> BlockReduceT;
  // Shared memory
  __shared__ typename BlockReduceT::TempStorage sdata;

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  Dist dst, min;

  dst = distances[tid];
  if (g_min->i == dst.i || g_min->i == dst.j || g_min->j == dst.i || g_min->j == dst.j) {
    if (tid >= distances_array_size || dst.j >= num_particles || dst.i >= num_particles) {
      dst.d = MAX_DOUBLE;
    } else if (dst.i == dst.j) {
      dst.d = jets[dst.i].get_diB(scheme);
    } else {
      dst.d = yij_distance(jets[dst.i], jets[dst.j], scheme, one_over_r2);
    }

    distances[tid] = dst;
  } else {
    if (tid >= distances_array_size || dst.j >= num_particles || dst.i >= num_particles) {
      dst.d = MAX_DOUBLE;
    }
  }

  // printf("%4d%4d%4d%4d%20.8e\n", blockIdx.x, num_particles, dst.i, dst.j, dst.d);
  min = BlockReduceT(sdata).Reduce(dst, dist_compare());

  if (threadIdx.x == 0) {
    distances_out[blockIdx.x] = min;
    // printf("%4d%4d%4d%4d%20.8e\n", blockIdx.x, num_particles, min.i, min.j, min.d);
  }
}

__global__ void reduction_blocks(
    PseudoJetExt *jets, Dist *distances_out, Dist *g_min, int const distances_array_size, int const num_particles) {
  // Specialize BlockReduce type for our thread block
  typedef BlockReduce<Dist, 1024> BlockReduceT;
  // Shared memory
  __shared__ typename BlockReduceT::TempStorage sdata;

  unsigned int tid = threadIdx.x;

  Dist dst;

  if (tid >= distances_array_size) {
    dst.d = MAX_DOUBLE;
  } else {
    dst = distances_out[tid];
    for (int i = tid + blockDim.x; i < distances_array_size; i += blockDim.x) {
      Dist temp = distances_out[i];
      if (temp.d < dst.d) {
        dst = temp;
      }
    }
  }

  Dist min = BlockReduceT(sdata).Reduce(dst, dist_compare());

  if (tid == 0) {
    (*g_min) = min;
    int i, j;
    i = min.i;
    j = min.j;

    //printf("%6d%6d%6d%20.8e\n", num_particles, i, j, min.d);

    // int f, e;
    // tid_to_ij(f, e, 58101);
    // printf("%4d%4d%4d%20.8e\n", num_particles, f, e, distances[58101]);

    if (i == j) {
      PseudoJetExt temp;
      temp = jets[j];
      jets[j] = jets[num_particles - 1];
      temp.isJet = true;
      jets[num_particles - 1] = temp;
    } else {
      jets[i].px += jets[j].px;
      jets[i].py += jets[j].py;
      jets[i].pz += jets[j].pz;
      jets[i].E += jets[j].E;
      _set_jet(jets[i]);

      jets[j] = jets[num_particles - 1];
    }
  }
}

__global__ void init(const PseudoJet *particles, PseudoJetExt *jets, int size) {
  int first = threadIdx.x + blockIdx.x * blockDim.x;
  int grid = blockDim.x * gridDim.x;

  for (int i = first; i < size; i += grid) {
    jets[i].px = particles[i].px;
    jets[i].py = particles[i].py;
    jets[i].pz = particles[i].pz;
    jets[i].E = particles[i].E;
    _set_jet(jets[i]);
  }
}

__global__ void output(const PseudoJetExt *jets, PseudoJet *particles, int size) {
  int first = threadIdx.x + blockIdx.x * blockDim.x;
  int grid = blockDim.x * gridDim.x;

  for (int i = first; i < size; i += grid) {
    particles[i].px = jets[i].px;
    particles[i].py = jets[i].py;
    particles[i].pz = jets[i].pz;
    particles[i].E = jets[i].E;
    particles[i].index = i;
    particles[i].isJet = jets[i].isJet;
  }
}

void cluster(PseudoJet *particles, int size, Scheme scheme, double r) {
#pragma regoin CudaMalloc
  PseudoJetExt *d_jets;
  cudaCheck(hipMalloc(&d_jets, size * sizeof(PseudoJetExt)));
  init<<<8, 512>>>(particles, d_jets, size);

  Dist *d_distances = 0;
  cudaCheck(hipMalloc((void **)&d_distances, size * (size + 1) / 2 * sizeof(Dist)));

  Dist *d_out = 0;
  cudaCheck(hipMalloc((void **)&d_out, size * sizeof(Dist)));

  Dist *d_min = 0;
  cudaCheck(hipMalloc((void **)&d_min, sizeof(Dist)));

#pragma endregoin

  int num_threads = size;
  int num_blocks = (size + num_threads) / (num_threads + 1);
  double one_over_r2 = 1. / (r * r);

  // Compute dIB, eta, phi for each jet
  set_jets<<<num_blocks, num_threads>>>(d_jets);

  // Compute distances
  num_threads = (size * (size + 1) / 2);
  num_blocks = (num_threads / 1024) + 1;
  set_distances<<<num_blocks, 1024>>>(d_jets, d_distances, d_min, num_threads, scheme, one_over_r2);

  // Loop n times reduce + recombine
  for (int n = size; n > 0; n--) {
    num_threads = (n * (n + 1) / 2);
    num_blocks = (num_threads / 1024) + 1;

    // Find the minimum in each block for the distances array
    reduction_min<<<num_blocks, 1024, 1024 * sizeof(Dist)>>>(
        d_jets, d_distances, d_out, d_min, num_threads, n, scheme, one_over_r2);

    // // Find the minimum of all blocks
    int b = upper_power_of_two(num_blocks - 1) + 1;
    // cout << num_blocks << "\t" << b + 1 << endl;
    reduction_blocks<<<1, 1024, 1024 * sizeof(Dist)>>>(d_jets, d_out, d_min, num_blocks, n);
  }

  output<<<8, 512>>>(d_jets, particles, size);

  cudaCheck(hipFree(d_jets));
  cudaCheck(hipFree(d_out));
  cudaCheck(hipFree(d_distances));
  cudaCheck(hipFree(d_min));
}
