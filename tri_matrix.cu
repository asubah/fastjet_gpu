#include "hip/hip_runtime.h"
#include <assert.h>
#include <cmath>
#include <cub/block/block_reduce.cuh>
#include <iostream>
#include <limits>
#include <numeric>
#include <stdio.h>
#include <vector>
// Here you can set the device ID that was assigned to you
#define MYDEVICE 0
#define OUTPUT_JETS false
#define BENCH !OUTPUT_JETS

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

unsigned int upper_power_of_two(int v) {

  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;

  return v;
}

using namespace std;
using namespace hipcub;

struct PseudoJet {
  double px;
  double py;
  double pz;
  double E;
  double diB;
  double phi;
  double rap;
  bool isJet;

  __host__ __device__ double get_diB() const {
    return diB > 1e-300 ? 1.0 / diB : 1e300;
  }
};

const double pi = 3.141592653589793238462643383279502884197;
const double twopi = 6.283185307179586476925286766559005768394;
const double MaxRap = 1e5;
const double R = 0.4;
const double R2 = R * R;
const double invR2 = 1.0 / R2;
const double MAX_DOUBLE = 1.79769e+308;
#if OUTPUT_JETS
const double ptmin = 1.0;
const double dcut = ptmin * ptmin;
#endif

__device__ void _set_jet(PseudoJet &jet) {
  jet.diB = jet.px * jet.px + jet.py * jet.py;
  jet.isJet = false;

  if (jet.diB == 0.0) {
    jet.phi = 0.0;
  } else {
    jet.phi = atan2(jet.py, jet.px);
  }
  if (jet.phi < 0.0) {
    jet.phi += twopi;
  }
  if (jet.phi >= twopi) {
    jet.phi -= twopi;
  } // can happen if phi=-|eps<1e-15|?
  if (jet.E == abs(jet.pz) && jet.diB == 0) {
    // Point has infinite rapidity -- convert that into a very large
    // number, but in such a way that different 0-pt momenta will have
    // different rapidities (so as to lift the degeneracy between
    // them) [this can be relevant at parton-level]
    double MaxRapHere = MaxRap + abs(jet.pz);
    if (jet.pz >= 0.0) {
      jet.rap = MaxRapHere;
    } else {
      jet.rap = -MaxRapHere;
    }
  } else {
    // get the rapidity in a way that's modestly insensitive to roundoff
    // error when things pz,E are large (actually the best we can do without
    // explicit knowledge of mass)
    double effective_m2 = max(0.0, (jet.E + jet.pz) * (jet.E - jet.pz) -
                                       jet.diB); // force non tachyonic mass
    double E_plus_pz = jet.E + abs(jet.pz);      // the safer of p+, p-
    // p+/p- = (p+ p-) / (p-)^2 = (kt^2+m^2)/(p-)^2
    jet.rap = 0.5 * log((jet.diB + effective_m2) / (E_plus_pz * E_plus_pz));
    if (jet.pz > 0) {
      jet.rap = -jet.rap;
    }
  }
}

__device__ double plain_distance(PseudoJet &jet1, PseudoJet &jet2) {
  double dphi = abs(jet1.phi - jet2.phi);
  if (dphi > pi) {
    dphi = twopi - dphi;
  }
  double drap = jet1.rap - jet2.rap;
  return (dphi * dphi + drap * drap);
}

__device__ double yij_distance(PseudoJet &jet1, PseudoJet &jet2) {
  return min(jet1.get_diB(), jet2.get_diB()) * plain_distance(jet1, jet2) *
         invR2;
}

__device__ void tid_to_ij(int &i, int &j, int tid) {
  tid += 1;
  j = std::ceil(std::sqrt(2 * tid + 0.25) - 0.5);
  i = trunc(tid - (j - 1) * j / 2.0);
  j -= 1;
  i -= 1;
}

__global__ void set_jets(PseudoJet *jets) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  _set_jet(jets[tid]);
}

struct Dist {
  double d;
  int i;
  int j;
};

struct dist_compare {
  __host__ __device__ Dist operator()(Dist &first, Dist &second) {
    return first.d < second.d ? first : second;
  }
};

__global__ void reduction_min(PseudoJet *jets, Dist *distances_out,
                              int const distances_array_size,
                              int const num_particles) {
  // Specialize BlockReduce type for our thread block
  typedef BlockReduce<Dist, 1024> BlockReduceT;
  // Shared memory
  __shared__ typename BlockReduceT::TempStorage sdata;

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  Dist dst;
  tid_to_ij(dst.i, dst.j, tid);

  if (tid >= distances_array_size || dst.j >= num_particles ||
      dst.i >= num_particles) {
    dst.d = MAX_DOUBLE;
  } else if (dst.i == dst.j) {
    dst.d = jets[dst.i].get_diB();
  } else {
    dst.d = yij_distance(jets[dst.i], jets[dst.j]);
  }

  Dist min = BlockReduceT(sdata).Reduce(dst, dist_compare());

  if (threadIdx.x == 0) {
    distances_out[blockIdx.x] = min;
    // printf("%4d%4d%4d%20.8e\n", num_particles, min.i, min.j, min.d);
  }
}

__global__ void reduction_blocks(PseudoJet *jets, Dist *distances_out,
                                 int const distances_array_size,
                                 int const num_particles) {
  // Specialize BlockReduce type for our thread block
  typedef BlockReduce<Dist, 512> BlockReduceT;
  // Shared memory
  __shared__ typename BlockReduceT::TempStorage sdata;

  unsigned int tid = threadIdx.x;

  Dist dst;

  if (tid >= distances_array_size) {
    dst.d = MAX_DOUBLE;
  } else {
    dst = distances_out[tid];
  }

  Dist min = BlockReduceT(sdata).Reduce(dst, dist_compare());

  if (tid == 0) {
    int i, j;
    i = min.i;
    j = min.j;

    // printf("block %4d%4d%4d%20.8e\n", num_particles, i, j, min.d);

    // int f, e;
    // tid_to_ij(f, e, 58101);
    // printf("%4d%4d%4d%20.8e\n", num_particles, f, e, distances[58101]);

    if (i == j) {
      PseudoJet temp;
      temp = jets[j];
      jets[j] = jets[num_particles - 1];
      temp.isJet = true;
      jets[num_particles - 1] = temp;
    } else {
      jets[i].px += jets[j].px;
      jets[i].py += jets[j].py;
      jets[i].pz += jets[j].pz;
      jets[i].E += jets[j].E;
      _set_jet(jets[i]);

      jets[j] = jets[num_particles - 1];
    }
  }
}

int main() {
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  printf("Device Name: %s\n", prop.name);

  int num_particles = 0;
  // Increase the number to process more events
  int num_events = 1;

  // Loop events
  for (int event = 0; event < num_events; event++) {
    PseudoJet *h_jets = NULL;
    PseudoJet *h_more_jets = NULL;
    PseudoJet temp;

    // Read particles
    num_particles = 0;
    while (true) {
      cin >> temp.px >> temp.py >> temp.pz >> temp.E;

      if (cin.fail())
        break;

      num_particles++;

      h_more_jets =
          (PseudoJet *)realloc(h_jets, num_particles * sizeof(PseudoJet));

      if (h_more_jets != NULL) {
        h_jets = h_more_jets;
        h_jets[num_particles - 1] = temp;
      } else {
        free(h_jets);
        puts("Error (re)allocating memory");
        exit(1);
      }
    }

    cin.clear();
    cin.ignore(numeric_limits<streamsize>::max(), '\n');

    // Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

#pragma regoin CudaMalloc
    PseudoJet *d_jets = 0;
    hipMalloc((void **)&d_jets, num_particles * sizeof(PseudoJet));
    hipMemcpy(d_jets, h_jets, num_particles * sizeof(PseudoJet),
               hipMemcpyHostToDevice);

    int num_threads = num_particles;
    int num_blocks = (num_particles + num_threads) / (num_threads + 1);

    Dist *d_out = 0;
    hipMalloc((void **)&d_out, num_threads * sizeof(Dist));
#pragma endregoin

// Benchmarking
#if BENCH
    float milliseconds;
    vector<double> acc;
    for (int s = 0; s < 1000; s++) {
      hipEventRecord(start);
#endif
      // Compute dIB, eta, phi for each jet
      set_jets<<<num_blocks, num_threads>>>(d_jets);

      // Loop n times reduce + recombine
      for (int n = num_particles; n > 0; n--) {
        num_threads = (n * (n + 1) / 2);
        num_blocks = (num_threads / 1024) + 1;

        // Find the minimum in each block for the distances array
        reduction_min<<<num_blocks, 1024, 1024 * sizeof(Dist)>>>(
            d_jets, d_out, num_threads, n);

        // // Find the minimum of all blocks
        int b = upper_power_of_two(num_blocks - 1) + 1;
        // cout << num_blocks << "\t" << b + 1 << endl;
        reduction_blocks<<<1, 512, 512 * sizeof(Dist)>>>(d_jets, d_out, num_blocks,
                                                     n);
      }
#if BENCH
      hipEventRecord(stop);
      hipEventSynchronize(stop);

      hipEventElapsedTime(&milliseconds, start, stop);
      printf("run %d\t%.3fms\n", s, milliseconds);
      acc.push_back(milliseconds);
    }

    double sum = std::accumulate(acc.begin(), acc.end(), 0.0);
    double mean = sum / acc.size();

    double sq_sum =
        std::inner_product(acc.begin(), acc.end(), acc.begin(), 0.0);
    double stdev = std::sqrt(sq_sum / acc.size() - mean * mean);
    printf("n =  %d\n", num_particles);
    printf("mean %.3fms\n", mean);
    printf("std %.3fms\n", stdev);
#endif

    checkCUDAError("kernal launch");

#if OUTPUT_JETS
    hipMemcpy(h_jets, d_jets, num_particles * sizeof(PseudoJet),
               hipMemcpyDeviceToHost);

    for (int i = 0; i < num_particles; i++)
      if (h_jets[i].diB >= dcut && h_jets[i].isJet)
        printf("%15.8f %15.8f %15.8f %15.8f\n",
               h_jets[i].px, h_jets[i].py, h_jets[i].pz, h_jets[i].E
               );
#endif

    // free device memory
    hipFree(d_jets);
    // hipFree(d_distances);
    // hipFree(d_indices);
    // hipFree(d_indices_ii);
    // hipFree(d_indices_jj);
    hipFree(d_out);

    free(h_jets);
  }

  return 0;
}

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(-1);
  }
}