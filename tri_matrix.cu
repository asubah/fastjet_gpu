
#include <hip/hip_runtime.h>
#include <assert.h>
#include <cmath>
#include <iostream>
#include <limits>
#include <numeric>
#include <stdio.h>
#include <vector>
// Here you can set the device ID that was assigned to you
#define MYDEVICE 0
#define OUTPUT_JETS false
#define BENCH true

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

using namespace std;

struct PseudoJet {
  double px;
  double py;
  double pz;
  double E;
  double diB;
  double phi;
  double rap;
  bool isJet;
};

const double pi = 3.141592653589793238462643383279502884197;
const double twopi = 6.283185307179586476925286766559005768394;
const double MaxRap = 1e5;
const double R = 0.6;
const double R2 = R * R;
const double invR2 = 1.0 / R2;
#if OUTPUT_JETS
const double ptmin = 5.0;
const double dcut = ptmin * ptmin;
#endif

__device__ void _set_jet(PseudoJet &jet) {
  jet.diB = jet.px * jet.px + jet.py * jet.py;
  jet.isJet = false;

  if (jet.diB == 0.0) {
    jet.phi = 0.0;
  } else {
    jet.phi = atan2(jet.py, jet.px);
  }
  if (jet.phi < 0.0) {
    jet.phi += twopi;
  }
  if (jet.phi >= twopi) {
    jet.phi -= twopi;
  } // can happen if phi=-|eps<1e-15|?
  if (jet.E == abs(jet.pz) && jet.diB == 0) {
    // Point has infinite rapidity -- convert that into a very large
    // number, but in such a way that different 0-pt momenta will have
    // different rapidities (so as to lift the degeneracy between
    // them) [this can be relevant at parton-level]
    double MaxRapHere = MaxRap + abs(jet.pz);
    if (jet.pz >= 0.0) {
      jet.rap = MaxRapHere;
    } else {
      jet.rap = -MaxRapHere;
    }
  } else {
    // get the rapidity in a way that's modestly insensitive to roundoff
    // error when things pz,E are large (actually the best we can do without
    // explicit knowledge of mass)
    double effective_m2 = max(0.0, (jet.E + jet.pz) * (jet.E - jet.pz) -
                                       jet.diB); // force non tachyonic mass
    double E_plus_pz = jet.E + abs(jet.pz);      // the safer of p+, p-
    // p+/p- = (p+ p-) / (p-)^2 = (kt^2+m^2)/(p-)^2
    jet.rap = 0.5 * log((jet.diB + effective_m2) / (E_plus_pz * E_plus_pz));
    if (jet.pz > 0) {
      jet.rap = -jet.rap;
    }
  }
}

__device__ double plain_distance(PseudoJet &jet1, PseudoJet &jet2) {
  double dphi = abs(jet1.phi - jet2.phi);
  if (dphi > pi) {
    dphi = twopi - dphi;
  }
  double drap = jet1.rap - jet2.rap;
  return (dphi * dphi + drap * drap);
}

__device__ double yij_distance(PseudoJet &jet1, PseudoJet &jet2) {
  return min(jet1.diB, jet2.diB) * plain_distance(jet1, jet2) * invR2;
}

__device__ void tid_to_ij(int &i, int &j, int tid) {
  tid += 1;
  j = std::ceil(std::sqrt(2 * tid + 0.25) - 0.5);
  i = trunc(tid - (j - 1) * j / 2.0);
  j -= 1;
  i -= 1;
}

__global__ void set_jets(PseudoJet *jets) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  _set_jet(jets[tid]);
}

__global__ void set_distances(PseudoJet *jets, double *distances, int *indices,
                              int *indices_ii, int *indices_jj,
                              int const num_particels) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int N = num_particels * (num_particels + 1) / 2;

  if (tid >= N)
    return;

  indices[tid] = tid;

  int i, j;
  tid_to_ij(i, j, tid);
  indices_ii[tid] = i;
  indices_jj[tid] = j;

  if (i == j) {
    distances[tid] = jets[i].diB;
  } else {
    distances[tid] = yij_distance(jets[i], jets[j]);
  }
}

__global__ void reduction_min(PseudoJet *jets, double *distances,
                              double *distances_out, int *indices,
                              int *indices_ii, int *indices_jj,
                              int const distances_array_size,
                              int const num_particels) {
  extern __shared__ double sdata[];
  double *s_distances = sdata;
  int *s_indices = (int *)&s_distances[blockDim.x];

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int i = threadIdx.x;

  if (tid >= distances_array_size)
    return;

  s_distances[i] = distances[tid];
  s_indices[i] = tid;
  __syncthreads();

  int jj;
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (i < s && (tid + s) < distances_array_size) {
      // tid_to_ij(ii, jj, s_indices[i + s]);
      // ii = indices_ii[s_indices[i + s]];
      jj = indices_jj[s_indices[i + s]];
      if (s_distances[i] > s_distances[i + s] && jj < num_particels) {
        s_distances[i] = s_distances[i + s];

        s_indices[i] = s_indices[i + s];
      }
    }
    __syncthreads();
  }

  if (i == 0) {
    distances_out[blockIdx.x] = s_distances[0];
    int min_tid = s_indices[0];
    indices[blockIdx.x] = min_tid;
  }
}

__global__ void reduction_blocks(PseudoJet *jets, double *distances,
                                 double *distances_out, int *indices,
                                 int *indices_ii, int *indices_jj,
                                 int const distances_array_size,
                                 int const num_particles) {
  extern __shared__ double sdata[];
  double *s_distances = sdata;
  int *s_indices = (int *)&s_distances[blockDim.x];

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int i = threadIdx.x;

  if (tid >= distances_array_size)
    return;

  s_distances[i] = distances[tid];
  s_indices[i] = indices[tid];
  __syncthreads();

  int jj;
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (i < s && (tid + s) < distances_array_size) {
      // tid_to_ij(ii, jj, s_indices[i + s]);
      // ii = indices_ii[s_indices[i + s]];
      jj = indices_jj[s_indices[i + s]];
      if (s_distances[i] > s_distances[i + s] && jj < num_particles) {
        s_distances[i] = s_distances[i + s];

        s_indices[i] = s_indices[i + s];
      }
    }
    __syncthreads();
  }

  if (i == 0) {
    distances_out[blockIdx.x] = s_distances[0];
    int min_tid = s_indices[0];
    indices[blockIdx.x] = min_tid;

    int i, j;
    // tid_to_ij(i, j, min_tid);
    i = indices_ii[min_tid];
    j = indices_jj[min_tid];

    if (i == j) {
      PseudoJet temp;
      temp = jets[j];
      jets[j] = jets[num_particles - 1];
      temp.isJet = true;
      jets[num_particles - 1] = temp;
    } else {
      jets[i].px += jets[j].px;
      jets[i].py += jets[j].py;
      jets[i].pz += jets[j].pz;
      jets[i].E += jets[j].E;
      _set_jet(jets[i]);

      jets[j] = jets[num_particles - 1];
    }
  }
}

__global__ void recombine_recalculate(PseudoJet *jets, double *distances,
                                      double *distances_out, int *indices,
                                      int *indices_ii, int *indices_jj,
                                      int const n, int d_out_size) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int i, j;
  int index = indices[0];

  // tid_to_ij(i, j, index);
  i = indices_ii[index];
  j = indices_jj[index];

  if (j >= n) {
    // tid_to_ij(i, j, index - n);
    i = indices_ii[index - n];
    j = indices_jj[index - n];
  }
  int tid_j = tid + ((j) * (j + 1) / 2);

  if (tid >= n)
    return;

  if (tid == j) {
    distances[tid_j] = jets[tid].diB;
  } else if (tid < j) {
    distances[tid_j] = yij_distance(jets[tid], jets[j]);
  } else {
    tid_j = j + ((tid) * (tid + 1) / 2);
    distances[tid_j] = yij_distance(jets[tid], jets[j]);
  }

  int tid_i = tid_j;
  if (i != j) {
    tid_i = tid + ((i) * (i + 1) / 2);
    if (tid == i)
      distances[tid_i] = jets[tid].diB;
    else if (tid < i) {
      distances[tid_i] = yij_distance(jets[tid], jets[i]);
    } else {
      tid_i = i + ((tid) * (tid + 1) / 2);
      distances[tid_i] = yij_distance(jets[tid], jets[i]);
    }
  }
}

int main() {
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  printf("Device Name: %s\n", prop.name);

  int num_particels = 0;
  // Increase the number to process more events
  int num_events = 1;

  // Loop events
  for (int event = 0; event < num_events; event++) {
    PseudoJet *h_jets = NULL;
    PseudoJet *h_more_jets = NULL;
    PseudoJet temp;

    // Read particles
    num_particels = 0;
    while (true) {
      cin >> temp.px >> temp.py >> temp.pz >> temp.E;

      if (cin.fail())
        break;

      num_particels++;

      h_more_jets =
          (PseudoJet *)realloc(h_jets, num_particels * sizeof(PseudoJet));

      if (h_more_jets != NULL) {
        h_jets = h_more_jets;
        h_jets[num_particels - 1] = temp;
      } else {
        free(h_jets);
        puts("Error (re)allocating memory");
        exit(1);
      }
    }

    cin.clear();
    cin.ignore(numeric_limits<streamsize>::max(), '\n');

    // Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

#pragma regoin CudaMalloc
    PseudoJet *d_jets = 0;
    hipMalloc((void **)&d_jets, num_particels * sizeof(PseudoJet));
    hipMemcpy(d_jets, h_jets, num_particels * sizeof(PseudoJet),
               hipMemcpyHostToDevice);

    double *d_distances = 0;
    hipMalloc((void **)&d_distances,
               (num_particels * (num_particels + 1) / 2) * sizeof(double));

    int *d_indices = 0;
    hipMalloc((void **)&d_indices,
               (num_particels * (num_particels + 1) / 2) * sizeof(int));
    int *d_indices_ii = 0;
    hipMalloc((void **)&d_indices_ii,
               (num_particels * (num_particels + 1) / 2) * sizeof(int));
    int *d_indices_jj = 0;
    hipMalloc((void **)&d_indices_jj,
               (num_particels * (num_particels + 1) / 2) * sizeof(int));

    int num_threads = 354;
    int num_blocks = (num_particels + num_threads) / (num_threads + 1);

    double *d_out = 0;
    hipMalloc((void **)&d_out, num_threads * sizeof(double));
#pragma endregoin

// Benchmarking
#if BENCH
    float milliseconds;
    vector<double> acc;
    for (int s = 0; s < 1000; s++) {
      hipEventRecord(start);
#endif
      // Compute dIB, eta, phi for each jet
      set_jets<<<num_blocks, num_threads>>>(d_jets);

      num_threads = (num_particels * (num_particels + 1) / 2);
      num_blocks = (num_threads / 1024) + 1;
      // Set an N * (N+1) / 2 array to store distances
      set_distances<<<num_blocks, 1024>>>(d_jets, d_distances, d_indices,
                                          d_indices_ii, d_indices_jj,
                                          num_particels);

      // Loop n times reduce + recombine
      for (int n = num_particels; n > 0; n--) {
        num_threads = (n * (n + 1) / 2);
        num_blocks = (num_threads / 1024) + 1;

        // Find the minimum in each block for the distances array
        reduction_min<<<num_blocks, 1024,
                        1024 * sizeof(double) + 1024 * sizeof(int)>>>(
            d_jets, d_distances, d_out, d_indices, d_indices_ii, d_indices_jj,
            num_threads, n);

        // Find the minimum of all blocks
        reduction_blocks<<<1, num_blocks, num_blocks * sizeof(double) +
                                              num_blocks * sizeof(int)>>>(
            d_jets, d_out, d_out, d_indices, d_indices_ii, d_indices_jj,
            num_blocks, n);

        // Find the minimum of all blocks + recombine + recalculate distances
        recombine_recalculate<<<(num_particels / 1024) + 1, 1024>>>(
            d_jets, d_distances, d_out, d_indices, d_indices_ii, d_indices_jj,
            n - 1, num_blocks);
      }
#if BENCH
      hipEventRecord(stop);
      hipEventSynchronize(stop);

      hipEventElapsedTime(&milliseconds, start, stop);
      printf("run %d\t%.3fms\n", s, milliseconds);
      acc.push_back(milliseconds);
    }

    double sum = std::accumulate(acc.begin(), acc.end(), 0.0);
    double mean = sum / acc.size();

    double sq_sum =
        std::inner_product(acc.begin(), acc.end(), acc.begin(), 0.0);
    double stdev = std::sqrt(sq_sum / acc.size() - mean * mean);
    printf("n =  %d\n", num_particels);
    printf("mean %.3fms\n", mean);
    printf("std %.3fms\n", stdev);
#endif

    checkCUDAError("kernal launch");

#if OUTPUT_JETS
    cudaMemcpy(h_jets, d_jets, num_particels * sizeof(PseudoJet),
               cudaMemcpyDeviceToHost);

    for (int i = 0; i < num_particels; i++)
      if (h_jets[i].diB >= dcut && h_jets[i].isJet)
        printf("%15.8f %15.8f %15.8f\n", h_jets[i].rap, h_jets[i].phi,
               sqrt(h_jets[i].diB));
#endif

    // free device memory
    hipFree(d_jets);
    hipFree(d_distances);
    hipFree(d_indices);
    hipFree(d_indices_ii);
    hipFree(d_indices_jj);
    hipFree(d_out);

    free(h_jets);
  }

  return 0;
}

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(-1);
  }
}