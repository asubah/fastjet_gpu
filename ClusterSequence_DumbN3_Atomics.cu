
#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>
#include <cmath>
#include <assert.h>

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

using namespace std;

struct PseudoJet
{
    double px;
    double py;
    double pz;
    double E;
    double diB;
    double phi;
    double rap;
};

const double pi = 3.141592653589793238462643383279502884197;
const double twopi = 6.283185307179586476925286766559005768394;
const double MaxRap = 1e5;
const double R = 0.6;
const double R2 = R * R;
const double invR2 = 1.0 / R2;
const double ptmin = 5.0;
const double dcut = ptmin * ptmin;
int const NUM_PARTICLES = 354;

__device__ static double atomicMin(double *address, double val)
{
    unsigned long long int *address_as_i = (unsigned long long int *)address;
    unsigned long long int old = *address_as_i, assumed;
    do
    {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __double_as_longlong(fmin(val, __longlong_as_double(assumed))));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__device__ void _set_jet(PseudoJet &jet)
{
    jet.diB = jet.px * jet.px + jet.py * jet.py;

    if (jet.diB == 0.0)
    {
        jet.phi = 0.0;
    }
    else
    {
        jet.phi = atan2(jet.py, jet.px);
    }
    if (jet.phi < 0.0)
    {
        jet.phi += twopi;
    }
    if (jet.phi >= twopi)
    {
        jet.phi -= twopi;
    } // can happen if phi=-|eps<1e-15|?
    if (jet.E == abs(jet.pz) && jet.diB == 0)
    {
        // Point has infinite rapidity -- convert that into a very large
        // number, but in such a way that different 0-pt momenta will have
        // different rapidities (so as to lift the degeneracy between
        // them) [this can be relevant at parton-level]
        double MaxRapHere = MaxRap + abs(jet.pz);
        if (jet.pz >= 0.0)
        {
            jet.rap = MaxRapHere;
        }
        else
        {
            jet.rap = -MaxRapHere;
        }
    }
    else
    {
        // get the rapidity in a way that's modestly insensitive to roundoff
        // error when things pz,E are large (actually the best we can do without
        // explicit knowledge of mass)
        double effective_m2 = max(0.0, (jet.E + jet.pz) * (jet.E - jet.pz) - jet.diB); // force non tachyonic mass
        double E_plus_pz = jet.E + abs(jet.pz);                                        // the safer of p+, p-
        // p+/p- = (p+ p-) / (p-)^2 = (kt^2+m^2)/(p-)^2
        jet.rap = 0.5 * log((jet.diB + effective_m2) / (E_plus_pz * E_plus_pz));
        if (jet.pz > 0)
        {
            jet.rap = -jet.rap;
        }
    }
}

__device__ double plain_distance(PseudoJet &jet1, PseudoJet &jet2)
{
    double dphi = abs(jet1.phi - jet2.phi);
    if (dphi > pi)
    {
        dphi = twopi - dphi;
    }
    double drap = jet1.rap - jet2.rap;
    return (dphi * dphi + drap * drap);
}

__global__ void dumb_n3(PseudoJet *jets, int num_particles)
{
    __shared__ PseudoJet s_jets[NUM_PARTICLES];
    __shared__ double minimum;
    __shared__ int s_ii, s_jj;

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < num_particles)
    {
        s_jets[tid] = jets[tid];
        _set_jet(s_jets[tid]);
        if (tid == 0)
        {
            minimum = 10000000000.0;
            s_ii = -1;
            s_jj = -1;
        }
        __syncthreads();
    }

    // Loop on all particles
    double ymin;
    int ii;
    int jj;
    while (num_particles > 0 && tid < num_particles)
    {
        ii = -1;
        jj = -1;
        // Find minimum yiB
        ymin = s_jets[tid].diB;
        double old = atomicMin(&minimum, ymin);
        // printf("old =  %f\n", old);
        // printf("ymin =  %f\n", ymin);
        // printf("minimum = %f\n", minimum);
        __syncthreads();
        if (ymin == minimum)
        {
            s_ii = tid;
            printf("ii = %d\n", tid);
        }
        ymin = minimum;
        __syncthreads();

        double distance = 0;
        for (int j = tid + 1; j < num_particles; j++)
        {
            distance = min(s_jets[tid].diB, s_jets[j].diB) * plain_distance(s_jets[tid], s_jets[j]) * invR2;
            // if(tid == 0)
            //     printf("%.17e\n", distance);
            if (distance < ymin)
            {
                ymin = distance;
                ii = tid;
                jj = j;
            }
        }

        // Find minimum yiB
        if (jj > -1)
        {
            atomicMin(&minimum, ymin);
            __syncthreads();
            if (ymin == minimum)
            {
                s_ii = ii;
                s_jj = jj;
                //
                // printf("ymin =  %f\n", ymin);
                // printf("minimum = %f\n", minimum);
            }
        }
        __syncthreads();
        // Get the minimum from all blocks
        if (tid == 0)
        {
            // printf("ii = %d, jj = %d\n", s_ii, s_jj);__syncthreads();
            // printf("minimum = %f\n", minimum);
            minimum = 100000000000.0;
            if (s_jj > -1)
            {
                // Do yij recombination
                s_jets[s_ii].px += s_jets[s_jj].px;
                s_jets[s_ii].py += s_jets[s_jj].py;
                s_jets[s_ii].pz += s_jets[s_jj].pz;
                s_jets[s_ii].E += s_jets[s_jj].E;
                _set_jet(s_jets[s_ii]);

                s_jets[s_jj] = s_jets[num_particles - 1];
            }
            else
            {
                // Do yiB recombination
                if (s_jets[s_ii].diB >= dcut)
                    printf("%15.8f %15.8f %15.8f\n",
                           s_jets[s_ii].rap, s_jets[s_ii].phi, sqrt(s_jets[s_ii].diB));

                s_jets[s_ii] = s_jets[num_particles - 1];
            }
            s_ii = -1;
            s_jj = -1;
        }

        num_particles--;
        __syncthreads();
    }
}

int main()
{
    hipSetDevice(MYDEVICE);

    PseudoJet *h_jets = 0;
    h_jets = (PseudoJet *)malloc(NUM_PARTICLES * sizeof(PseudoJet));

    double *h_mini = 0;
    h_mini = (double *)malloc(sizeof(double));
    *h_mini = numeric_limits<double>::max();

    int i;
    for (i = 0; i < NUM_PARTICLES; i++)
    {
        cin >> h_jets[i].px >> h_jets[i].py >> h_jets[i].pz >> h_jets[i].E;
    }

    PseudoJet *d_jets = 0;
    hipMalloc((void **)&d_jets, NUM_PARTICLES * sizeof(PseudoJet));
    hipMemcpy(d_jets, h_jets, NUM_PARTICLES * sizeof(PseudoJet), hipMemcpyHostToDevice);

    double *d_mini = 0;
    hipMalloc((void **)&d_mini, sizeof(double));
    hipMemcpy(d_mini, h_mini, sizeof(double), hipMemcpyHostToDevice);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy calls1");

    int num_threads = 1024; //354;
    int num_blocks = (NUM_PARTICLES - 1) / num_threads + 1;
    //std::cout << "blocks = " << num_blocks;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    dumb_n3<<<num_blocks,
              num_threads //,
              //NUM_PARTICLES * sizeof(PseudoJet) // Jets
              //    + NUM_PARTICLES * sizeof(double) // Distances
              //    + NUM_PARTICLES * 2 * sizeof(int)
              >>>(d_jets, NUM_PARTICLES);
    hipEventRecord(stop);

    // Check for any CUDA errors
    checkCUDAError("kernal launch");

    hipMemcpy(h_jets, d_jets,
               NUM_PARTICLES * sizeof(PseudoJet),
               hipMemcpyDeviceToHost);
    // Check for any CUDA errors
    checkCUDAError("cudaMemcpy2 calls");

    hipMemcpy(h_mini, d_mini, sizeof(double), hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("cudaMemcpy3 calls");
    hipEventSynchronize(stop);

    //    cout << "d_mini = " << *h_mini << std::endl;
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time = %10.8f\n", milliseconds);

    // free device memory
    hipFree(d_jets);
    hipFree(d_mini);

    // free host memory
    free(h_jets);
    free(h_mini);

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
